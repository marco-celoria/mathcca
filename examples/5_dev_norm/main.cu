#include "hip/hip_runtime.h"
#include <mathcca/host_matrix.h>
#include <mathcca/device_matrix.h>
#include <mathcca/norm.h>
#include <mathcca/detail/norm_impl.h>
#include <iostream>
#include <mathcca/fill_rand.h>
#include <mathcca/fill_const.h>
#include <mathcca/copy.h>

int main(int argc, char **argv)  {
  constexpr std::size_t l{46504};
  constexpr std::size_t m{38333};
  constexpr std::size_t n{l * m};
#ifdef _USE_DOUBLE_PRECISION
  using value_type= double;
#else
  using value_type= float;
#endif
  mathcca::device_matrix<value_type> dA{l, m};

  mathcca::fill_rand(dA.begin(), dA.end());
  
  auto res= std::sqrt(static_cast<value_type>(n/3.));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  auto resB= mathcca::frobenius_norm(dA, mathcca::Norm::Base());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tB_ms;
  hipEventElapsedTime(&tB_ms, start, stop);

  std::cout << "\n" << "Does Base result agree with real result? " << resB << " -:- " << res << "\tError: " << std::abs(resB - res) << "\n";
  std::cout         << "Base  time: " << tB_ms << "\n";

#ifdef _CUBLAS
  hipEventRecord(start);
  auto resC= mathcca::frobenius_norm(dA, mathcca::Norm::Cublas());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tC_ms;
  hipEventElapsedTime(&tC_ms, start, stop);

  std::cout << "\n" << "Does Cublas result agree with real result? " << resC << " -:- " << res  << " \tError: " << std::abs(resC - res ) << "\n";
  std::cout         << "Does Cublas result agree with Base result? " << resC << " -:- " << resB << " \tError: " << std::abs(resC - resB) << "\n";
  std::cout << "Cublas  time: " << tC_ms << "\n";

#endif

#ifdef _THRUST
  hipEventRecord(start);
  auto resT= mathcca::detail::frobenius_norm(mathcca::Thrust(), dA.cbegin().get(), dA.cend().get(), mathcca::Norm::Base());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tT_ms;
  hipEventElapsedTime(&tT_ms, start, stop);

  std::cout << "\n" << "Does Thrust result agree with real result? " << resT << " -:- " << res  << " \tError: " << std::abs(resT - res ) << "\n";
  std::cout         << "Does Thrust result agree with Base result? " << resT << " -:- " << resB << " \tError: " << std::abs(resT - resB) << "\n";
  std::cout << "Thrust  time: " << tT_ms << "\n";

#endif

  // destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);

}


