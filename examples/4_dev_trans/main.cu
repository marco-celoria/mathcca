#include <mathcca/host_matrix.h>
#include <mathcca/device_matrix.h>
#include <mathcca/transpose.h>
#include <iostream>
#include <mathcca/fill_rand.h>
#include <mathcca/fill_const.h>
#include <mathcca/copy.h>

int main(int argc, char **argv)  {
  
  constexpr std::size_t l{33504};
  constexpr std::size_t m{34333};

#ifdef _USE_DOUBLE_PRECISION
  using value_type= double;
#else
  using value_type= float;
#endif

  mathcca::device_matrix<value_type> dA{l, m};

  mathcca::fill_rand(dA.begin(), dA.end());

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  auto dB0= mathcca::transpose(dA, mathcca::Trans::Base());
  auto dB1= mathcca::transpose(dB0, mathcca::Trans::Tiled());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tB_milliseconds;
  hipEventElapsedTime(&tB_milliseconds, start, stop);

  hipEventRecord(start);
  auto dT0= mathcca::transpose(dA, mathcca::Trans::Tiled());
  auto dT1= mathcca::transpose(dT0, mathcca::Trans::Tiled());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tT_milliseconds;
  hipEventElapsedTime(&tT_milliseconds, start, stop);
  
  std::cout << std::boolalpha << (dA  == dB1) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (dA  == dT1) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (dB0 == dT0) << std::noboolalpha << "\n";

  std::cout << "tB == " << tB_milliseconds << " ms\n";
  std::cout << "tT == " << tT_milliseconds << " ms\n";

#ifdef _CUBLAS
  hipEventRecord(start);
  auto dC0= mathcca::transpose(dA,  mathcca::Trans::Cublas());
  auto dC1= mathcca::transpose(dC0, mathcca::Trans::Cublas());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tC_milliseconds;
  hipEventElapsedTime(&tC_milliseconds, start, stop);
  
  std::cout << std::boolalpha << (dA  == dC1) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (dC0 == dB0) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (dC0 == dT0) << std::noboolalpha << "\n";
  
  std::cout << "tC == " << tC_milliseconds << " ms\n";

#endif

  // destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);

#ifdef _HOST_CHECK
  mathcca::host_matrix<value_type> hA{l,m};
  mathcca::host_matrix<value_type> hB0{m,l};
  mathcca::host_matrix<value_type> hT0{m,l};
  mathcca::copy(dA.cbegin(),  dA.cend(),  hA.begin());
  mathcca::copy(dB0.cbegin(), dB0.cend(), hB0.begin());
  mathcca::copy(dT0.cbegin(), dT0.cend(), hT0.begin());
  hipDeviceSynchronize();

  auto hD0= mathcca::transpose(hA,  mathcca::Trans::Tiled());
  auto hD1= mathcca::transpose(hD0, mathcca::Trans::Tiled());

  std::cout << std::boolalpha << (hA  == hD1) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (hD0 == hB0) << std::noboolalpha << "\n";
  std::cout << std::boolalpha << (hD0 == hT0) << std::noboolalpha << "\n";
#endif

}


