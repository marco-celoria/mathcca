#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: 2025 Marco Celoria <celoria.marco@gmail.com>
 *
 * SPDX-License-Identifier: AGPL-3.0-or-later
 */

#include <mathcca/host_matrix.h>
#include <mathcca/device_matrix.h>
#include <mathcca/norm.h>
#include <mathcca/detail/norm_impl.h>
#include <iostream>
#include <mathcca/fill_rand.h>
#include <mathcca/fill_const.h>
#include <mathcca/copy.h>

#ifdef _OPENMP
 #include <omp.h>
#endif

int main(int argc, char **argv)  {
  constexpr std::size_t l{46504};
  constexpr std::size_t m{38333};
  constexpr std::size_t n{l * m};

#ifdef _USE_DOUBLE_PRECISION
  std::cout << "USE DOUBLE PRECISION\n";
  using value_type= double;
#else
  std::cout << "USE SINGLE PRECISION\n";
  using value_type= float;
#endif

#ifdef _OPENMP
  int num_threads = 0;
  #pragma omp parallel reduction(+:num_threads)
  num_threads += 1;
  std::cout << "Running with " << num_threads << " OMP threads\n";
#endif

  mathcca::device_matrix<value_type> dA{l, m};

  mathcca::fill_rand(dA.begin(), dA.end());
  
  auto res= std::sqrt(static_cast<value_type>(n/3.));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  auto resB= mathcca::frobenius_norm(dA, mathcca::Norm::Base());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tB_ms;
  hipEventElapsedTime(&tB_ms, start, stop);

  std::cout << "\n" << "Does Base result agree with real result? " << resB << " -:- " << res << "\tError: " << std::abs(resB - res) << "\n";
  std::cout         << "Base  time: " << tB_ms << "\n";

#ifdef _CUBLAS
  hipEventRecord(start);
  auto resC= mathcca::frobenius_norm(dA, mathcca::Norm::Cublas());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tC_ms;
  hipEventElapsedTime(&tC_ms, start, stop);

  std::cout << "\n" << "Does Cublas result agree with real result? " << resC << " -:- " << res  << " \tError: " << std::abs(resC - res ) << "\n";
  std::cout         << "Does Cublas result agree with Base result? " << resC << " -:- " << resB << " \tError: " << std::abs(resC - resB) << "\n";
  std::cout << "Cublas  time: " << tC_ms << "\n";

#endif

#ifdef _THRUST
  hipEventRecord(start);
  auto resT= mathcca::detail::frobenius_norm(mathcca::Thrust(), dA.cbegin().get(), dA.cend().get(), mathcca::Norm::Base());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float tT_ms;
  hipEventElapsedTime(&tT_ms, start, stop);

  std::cout << "\n" << "Does Thrust result agree with real result? " << resT << " -:- " << res  << " \tError: " << std::abs(resT - res ) << "\n";
  std::cout         << "Does Thrust result agree with Base result? " << resT << " -:- " << resB << " \tError: " << std::abs(resT - resB) << "\n";
  std::cout << "Thrust  time: " << tT_ms << "\n";

#endif

  // destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);

}


