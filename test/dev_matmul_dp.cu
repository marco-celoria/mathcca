#include <mathcca.hpp>
#include <gtest/gtest.h>

TEST(MatmulDp, BasicAssertions)
{
    std::size_t l{5};
    std::size_t m{3};
    std::size_t n{2};
    for (auto i= 1; i < 8; ++i) {
      mathcca::device_matrix<double> dX0{l, m};
      mathcca::host_matrix<double>   hX0{l, m};
      mathcca::device_matrix<double> dY0{m, n};
      mathcca::host_matrix<double>   hY0{m, n};
      mathcca::device_matrix<double> dB0{l, n};
      mathcca::host_matrix<double>   hB0{l, n};
      mathcca::device_matrix<double> dT0{l, n};
      mathcca::host_matrix<double>   hT0{l, n};
      mathcca::device_matrix<double> dC0{l, n};
      mathcca::device_matrix<double> dERR{99, 99};
      mathcca::host_matrix<double>   hERR{99, 99};

      mathcca::fill_rand(dX0.begin(),  dX0.end());
      mathcca::copy(     dX0.cbegin(), dX0.cend(), hX0.begin());
      hipDeviceSynchronize();
      
      mathcca::fill_rand(dY0.begin(),  dY0.end());
      mathcca::copy(     dY0.cbegin(), dY0.cend(), hY0.begin());
      hipDeviceSynchronize();
      
      EXPECT_TRUE(dX0 != dY0);
      EXPECT_TRUE(hX0 != hY0);
      
      using value_type= typename decltype(dX0)::value_type;
      
      EXPECT_THROW({mathcca::matmul(dX0, dERR, mathcca::MM::Base());},  std::length_error);
      EXPECT_THROW({mathcca::matmul(hX0, hERR, mathcca::MM::Base());},  std::length_error);
      EXPECT_THROW({mathcca::matmul(dX0, dERR, mathcca::MM::Tiled());}, std::length_error);
      EXPECT_THROW({mathcca::matmul(hX0, hERR, mathcca::MM::Tiled());}, std::length_error);
      
      mathcca::matmul<value_type, mathcca::MM::Base, 8>(dX0, dY0, dB0, mathcca::MM::Base());
      mathcca::matmul<value_type, mathcca::MM::Base>   (hX0, hY0, hB0,  mathcca::MM::Base());
      auto dB1= mathcca::matmul<value_type, mathcca::MM::Base, 16>(dX0, dY0, mathcca::MM::Base());
      auto dB2= mathcca::matmul<value_type, mathcca::MM::Base, 32>(dX0, dY0, mathcca::MM::Base());
      auto hB1= mathcca::matmul<value_type, mathcca::MM::Base>    (hX0, hY0, mathcca::MM::Base());
      
      EXPECT_TRUE(dB0 == dB1);
      EXPECT_TRUE(dB1 == dB2);
      
      EXPECT_TRUE(hB0 == hB1);
      
      mathcca::matmul<value_type, mathcca::MM::Tiled, 8>(dX0, dY0, dT0, mathcca::MM::Tiled());
      mathcca::matmul<value_type, mathcca::MM::Tiled>   (hX0, hY0, hT0, mathcca::MM::Tiled());
      auto dT1= mathcca::matmul<value_type, mathcca::MM::Tiled, 16>(dX0, dY0, mathcca::MM::Tiled());
      auto dT2= mathcca::matmul<value_type, mathcca::MM::Tiled, 32>(dX0, dY0, mathcca::MM::Tiled());
      auto hT1= mathcca::matmul<value_type, mathcca::MM::Tiled>    (hX0, hY0, mathcca::MM::Tiled());
      
      EXPECT_TRUE(dT0 == dT1);
      EXPECT_TRUE(dT1 == dT2);
      
      EXPECT_TRUE(dT0 == dB0);
      EXPECT_TRUE(dT1 == dB1);
      EXPECT_TRUE(dT2 == dB2);
      
      EXPECT_TRUE(hT0 == hT1);
      EXPECT_TRUE(hT0 == hB0);
      EXPECT_TRUE(hT1 == hB1);

      auto dRB{dB0};
      auto dRT{dT0};

      mathcca::copy(hB0.cbegin(), hB0.cend(), dRB.begin());
     
      mathcca::copy(hT0.cbegin(), hT0.cend(), dRT.begin());

      EXPECT_TRUE(dB0 == dRB);
      EXPECT_TRUE(dT0 == dRT);

#ifdef _CUBLAS

      mathcca::matmul<value_type, mathcca::MM::Cublas, 8>(dX0, dY0, dC0, mathcca::MM::Cublas());
      auto dC1 = mathcca::matmul<value_type, mathcca::MM::Cublas, 16>(dX0, dY0, mathcca::MM::Cublas());
      auto dC2 = mathcca::matmul<value_type, mathcca::MM::Cublas, 32>(dX0, dY0, mathcca::MM::Cublas());
      
      EXPECT_TRUE(dC0 == dC1);
      EXPECT_TRUE(dC1 == dC2);

      EXPECT_TRUE(dC0 == dB0);
      EXPECT_TRUE(dC1 == dB1);
      EXPECT_TRUE(dC2 == dB2);

      EXPECT_TRUE(dC0 == dT0);
      EXPECT_TRUE(dC1 == dT1);
      EXPECT_TRUE(dC2 == dT2);      

#endif

      std::swap(l,n);
      l *= 5;
      m *= 3;
      n *= 2;
  }
}


